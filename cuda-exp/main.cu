
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
__global__ void add(int a, int b){
    int c = a + b;
    int d = c + a;
    printf("c: %d\n", c);
}

void test_add(int a, int b){
    add<<<1, 1>>>(a, b);
    hipDeviceReset(); // 同步设备函数的结果
}

int main() {
    int a = 1, b = 2;
    test_add(a, b);
    std::cout << "Hello, World!" << std::endl;
    return 0;
}
